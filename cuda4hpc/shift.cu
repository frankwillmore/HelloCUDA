/******************************************************************************/
/*                                                                            */
/*  (C) 2010 Texas Advanced Computing Center.  All rights reserved.           */
/*  For information, contact Frank Willmore:  willmore@tacc.utexas.edu        */
/*                                                                            */
/******************************************************************************/


#include <hip/hip_runtime.h>
#include <stdio.h>

__device__ char d_string[65536][256];

__global__ void toUpper() {
  if ((d_string[blockIdx.x][threadIdx.x] <= 122) && (d_string[blockIdx.x][threadIdx.x]) >=97)
    d_string[blockIdx.x][threadIdx.x] -= 32;
}

int main(int argc, char* argv[]) {
  char line[65536][256];
  int n_lines;

  for (n_lines=0; !feof(stdin); n_lines++) fgets(&line[n_lines][0], 256, stdin);

  hipMemcpyToSymbol(HIP_SYMBOL(d_string), line, sizeof(line), 0, hipMemcpyHostToDevice);
  toUpper<<< n_lines, 256 >>>();
  hipMemcpyFromSymbol(line, HIP_SYMBOL(d_string), sizeof(line), 0, hipMemcpyDeviceToHost);

  for (int i=0; i<n_lines; i++) printf("%s", line[i]);
}
